#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "mdsc/force.h"
#include "mdsc/vec.h"

extern "C" {

__global__ void lj_kernel(const double* __restrict__ rx,
                          const double* __restrict__ ry,
                          const double* __restrict__ rz,
                          double* __restrict__ fx,
                          double* __restrict__ fy,
                          double* __restrict__ fz,
                          double* __restrict__ pe_arr,
                          int n, double Lx, double Ly, double Lz, double rc2)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i>=n) return;
  double rxi = rx[i], ryi = ry[i], rzi = rz[i];
  double fxi=0.0, fyi=0.0, fzi=0.0;
  double pei=0.0;
  for (int j=0;j<n;++j){
    if (j==i) continue;
    double dx = rxi - rx[j];
    double dy = ryi - ry[j];
    double dz = rzi - rz[j];
    // minimum image
    if (dx >  0.5*Lx) dx -= Lx; if (dx < -0.5*Lx) dx += Lx;
    if (dy >  0.5*Ly) dy -= Ly; if (dy < -0.5*Ly) dy += Ly;
    if (dz >  0.5*Lz) dz -= Lz; if (dz < -0.5*Lz) dz += Lz;
    double r2 = dx*dx + dy*dy + dz*dz;
    if (r2 >= rc2 || r2==0.0) continue;
    double inv2 = 1.0/r2;
    double inv6 = inv2*inv2*inv2;
    double inv12 = inv6*inv6;
    double fscal = (48.0*inv12 - 24.0*inv6) * inv2;
    fxi += fscal*dx;
    fyi += fscal*dy;
    fzi += fscal*dz;
    // Potential (counted twice across i and j; we'll halve later on host)
    pei += 4.0*(inv6*inv6 - inv6);
  }
  fx[i] = fxi; fy[i] = fyi; fz[i] = fzi;
  pe_arr[i] = pei;
}

void lj_forces_cuda(system_t* s, force_accum_t* acc)
{
  int n = s->n;
  double *rx, *ry, *rz, *fx, *fy, *fz, *pe_arr;
  hipMalloc(&rx, n*sizeof(double));
  hipMalloc(&ry, n*sizeof(double));
  hipMalloc(&rz, n*sizeof(double));
  hipMalloc(&fx, n*sizeof(double));
  hipMalloc(&fy, n*sizeof(double));
  hipMalloc(&fz, n*sizeof(double));
  hipMalloc(&pe_arr, n*sizeof(double));

  double *hrx=(double*)malloc(n*sizeof(double));
  double *hry=(double*)malloc(n*sizeof(double));
  double *hrz=(double*)malloc(n*sizeof(double));
  for (int i=0;i<n;++i){ hrx[i]=s->p[i].r.x; hry[i]=s->p[i].r.y; hrz[i]=s->p[i].r.z; }

  hipMemcpy(rx, hrx, n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(ry, hry, n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(rz, hrz, n*sizeof(double), hipMemcpyHostToDevice);

  int block=128, grid=(n+block-1)/block;
  lj_kernel<<<grid,block>>>(rx,ry,rz,fx,fy,fz,pe_arr,n,s->box[0],s->box[1],s->box[2],s->rc2);
  hipDeviceSynchronize();

  double *hfx=(double*)malloc(n*sizeof(double));
  double *hfy=(double*)malloc(n*sizeof(double));
  double *hfz=(double*)malloc(n*sizeof(double));
  double *hpe=(double*)malloc(n*sizeof(double));
  hipMemcpy(hfx, fx, n*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(hfy, fy, n*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(hfz, fz, n*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(hpe, pe_arr, n*sizeof(double), hipMemcpyDeviceToHost);

  for (int i=0;i<n;++i){
    s->p[i].f.x = hfx[i];
    s->p[i].f.y = hfy[i];
    s->p[i].f.z = hfz[i];
  }
  // Sum potential (each pair counted twice in the simple O(N^2) kernel)
  double pe_sum = 0.0;
  for (int i=0;i<n;++i) pe_sum += hpe[i];
  acc->pe = 0.5 * pe_sum;

  free(hrx); free(hry); free(hrz);
  free(hfx); free(hfy); free(hfz);
  free(hpe);
  hipFree(rx); hipFree(ry); hipFree(rz);
  hipFree(fx); hipFree(fy); hipFree(fz);
  hipFree(pe_arr);
}

} // extern "C"
